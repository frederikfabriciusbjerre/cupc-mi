#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath> // For math functions
#include "pt.cuh"


// Example kernel that computes the CDF for an array of t-values
__global__ void compute_cdf(double *t_values, double *df_values, double *results, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        double t = t_values[idx];
        double df = df_values[idx];

        // Call the pt function
        double cdf = pt(t, df);

        // Store the result
        results[idx] = cdf;
    }
}

int main() {
    // Number of elements
    int n = 10;

    // Host arrays
    double *h_t_values = (double*)malloc(n * sizeof(double));
    double *h_df_values = (double*)malloc(n * sizeof(double));
    double *h_results = (double*)malloc(n * sizeof(double));

    // Initialize t-values and df-values
    for (int i = 0; i < n; i++) {
        h_t_values[i] = -3.0 + i * 0.7; // Example t-values from -3 to 3
        h_df_values[i] = 10.0;          // Degrees of freedom
    }

    // Device arrays
    double *d_t_values, *d_df_values, *d_results;
    hipMalloc((void**)&d_t_values, n * sizeof(double));
    hipMalloc((void**)&d_df_values, n * sizeof(double));
    hipMalloc((void**)&d_results, n * sizeof(double));

    // Copy data to device
    hipMemcpy(d_t_values, h_t_values, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_df_values, h_df_values, n * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    compute_cdf<<<gridSize, blockSize>>>(d_t_values, d_df_values, d_results, n);

    // Copy results back to host
    hipMemcpy(h_results, d_results, n * sizeof(double), hipMemcpyDeviceToHost);

    // Output the results
    for (int i = 0; i < n; i++) {
        printf("t: %f, df: %f, CDF: %f\n", h_t_values[i], h_df_values[i], (1-h_results[i]));
    }

    // Free memory
    hipFree(d_t_values);
    hipFree(d_df_values);
    hipFree(d_results);
    free(h_t_values);
    free(h_df_values);
    free(h_results);

    return 0;
}
